#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#define N 1024 //wielko�� obliczanych wektor�w
#define imin(a, b) (a<b?a:b)
const int threadsPerBlock = 256; //ilo�� w�tk�w na k�zdy blok
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);//ilo�� wykorzystywanych blok�w

__global__ void multiplyMatrix(float *a, float *b, float *c) {
	__shared__ float cache[threadsPerBlock]; //Zmienna dzielona ze wszystkimi w�tkami w tym bloku. Nie dzieli si� z innymi blokami!

	int tid = threadIdx.x + blockIdx.x * blockDim.x; //id w�tku kt�ry to wykonuje, id w�tku + id bloku * pojemno�� bolku
	int cacheIndex = threadIdx.x; //id cache, kt�re jest takie samo jak id obecnego w�tku 

	float temp = 0;
	while (tid < N) {
		temp = a[tid] * b[tid]; //zapis mno�enia w zmiennej 
		tid += blockDim.x * gridDim.x; //przesuwanie o ilo�� wszystkich w�tk�w w ca�ej siatce, nie trzeba ogarnia� na czw�rk� 
	}

	cache[cacheIndex] = temp; //przypisanie wyniku mno�enia do wsp�dzielonej tablicy cache

	__syncthreads(); //czekanie a� wszystkie w�tki dotr� to tego miejsca

	//tu troch� w powalony spos�b sumuj� si� wszystkie wyniki
	int i = blockDim.x / 2;
	while (i != 0) {
		if (cacheIndex < i) {
			cache[cacheIndex] += cache[cacheIndex + i];
		}

		__syncthreads();
		i /= 2;
	}
	//przypisanie sumy wszystkich wynik�w mno�enia do tablicy c
	if (cacheIndex == 0)
		c[blockIdx.x] = cache[0]; //jako, �e cache nie jest wsp�dzielony pomi�dzy blokami to wynik�w b�dzie tyle ile by�o wykorzystanych blok�w, p�niej to si� sumuje na cpu

}




int main(void) {
	float *a, *b, c, *partial_c; //deklarowanie tablic cpu
	float *dev_a, *dev_b, *dev_partial_c; //a tu gpu device

	//umieszczanie zmiennych w pami�ci CPU
	a = (float*)malloc(N * sizeof(float));
	b = (float*)malloc(N * sizeof(float));
	partial_c = (float*)malloc(blocksPerGrid * sizeof(float));
	//umieszczanie zmiennych w pami�ci GPU
	hipMalloc((void**)&dev_a, N * sizeof(float));
	hipMalloc((void**)&dev_b, N * sizeof(float));
	hipMalloc((void**)&dev_partial_c, blocksPerGrid * sizeof(float));
	//jakie� uzupe�nianie tablic z cpu
	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = 2 * i;
	}
	//kopiowanie zawarto�ci tablic z cpu do tablic na gpu
	hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);
	//wykonywanie funkcji na device, tu deklarujemy ilo�� wykorzystanych blok�w oraz w�tk�w na ka�dy blok (w obu zadaniach na 4 wystarczy�o <<<1, 1>>> jeden blok i jeden w�tek)
	multiplyMatrix << <blocksPerGrid, threadsPerBlock >> > (dev_a, dev_b, dev_partial_c);
	//kopiowanie wynik�w z gpu na cpu
	hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);
	//sumowanie wszystkich wynik�w z r�znych blok�w ju� musi si� wykona� na cpu
	c = 0;
	for (int i = 0; i < blocksPerGrid; i++) {
		c += partial_c[i];
	}

	printf("Matrix A: ");
	for (int i = 0; i < N; i++)
		printf("%.0f ", a[i]);
	printf("\nMatrix B: ");
	for (int i = 0; i < N; i++)
		printf("%.0f ", b[i]);
	printf("\nA * B: %.0f", c);
	//zwalnianie pami�ci z gpu
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_partial_c);
	//zwalnianie pami�ci z cpu
	free(a);
	free(b);
	free(partial_c);

	getchar();
	return 0;

}